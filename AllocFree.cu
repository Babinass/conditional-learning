/**********************************************************************
Code associated to the paper:

CONDITIONAL MONTE CARLO LEARNING FOR DIFFUSIONS

by: 

Lokman A. Abbas-Turki, Babacar Diallo and Giles Pagès

Those who re-use this code should mention in their code 
the name of the authors above.
**********************************************************************/

#include "Variables.h"
using namespace std;

////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
// Memory allocation for RNG 
////////////////////////////////////////////////////////////
void RngMalloc(void)
{
	testCUDA(hipMalloc(&pt_CMRG, sizeof(TabSeedCMRG_t)));
	testCUDA(hipMalloc(&pt_2RNG, sizeof(Tab2RNG_t)));
	testCUDA(hipHostAlloc(&pt_CMRGCPU, sizeof(TabSeedCMRG_t),
		hipHostMallocDefault));
}

////////////////////////////////////////////////////////////
// Memory deallocation for RNG 
////////////////////////////////////////////////////////////
void RngFree(void)
{
	testCUDA(hipFree(pt_CMRG));
	testCUDA(hipFree(pt_2RNG));
	testCUDA(hipHostFree(pt_CMRGCPU));
}

////////////////////////////////////////////////////////////
// Memory allocation for outer trajectories
////////////////////////////////////////////////////////////
void OutMalloc(AP ap)
{
	int NI = ap.NI;
	testCUDA(hipMalloc(&m11, 2*NbOuter * sizeof(float)));
	testCUDA(hipMalloc(&m12, 2*NbOuter * sizeof(float)));

	testCUDA(hipMalloc(&m21, NbOuter * sizeof(float)));
	testCUDA(hipMalloc(&m22, NbOuter * sizeof(float)));

	testCUDA(hipMalloc(&SIG, 8 * sizeof(float)));
	testCUDA(hipMalloc(&Alpha, NbOuter * sizeof(float)));

	testCUDA(hipMalloc(&X, (NI + 1)*Dim*NbOuter*sizeof(float)));

	testCUDA(hipMalloc(&Y, (NI+1)*NbOuter*sizeof(float)));
	testCUDA(hipMalloc(&Z, NI*Dim*NbOuter*sizeof(float)));

	testCUDA(hipMalloc(&val1, sizeof(float)));
	testCUDA(hipMalloc(&var1, sizeof(float)));
	testCUDA(hipMalloc(&val2, sizeof(float)));
	testCUDA(hipMalloc(&var2, sizeof(float)));		

	testCUDA(hipMalloc(&EZ, NI*Dim*sizeof(float)));
	testCUDA(hipMalloc(&VZ, NI*Dim*sizeof(float)));
	testCUDA(hipMalloc(&EZt, NI*Dim*sizeof(float)));
	testCUDA(hipMalloc(&VZt, NI*Dim*sizeof(float)));	
	testCUDA(hipMalloc(&EY, NI*sizeof(float)));
	testCUDA(hipMalloc(&VY, NI*sizeof(float)));	
	testCUDA(hipMalloc(&EYt, NI*sizeof(float)));
	testCUDA(hipMalloc(&VYt, NI*sizeof(float)));		
}

////////////////////////////////////////////////////////////
// Memory deallocation for outer trajectories
////////////////////////////////////////////////////////////
void OutFree(void)
{
	testCUDA(hipFree(m11));
	testCUDA(hipFree(m12));

	testCUDA(hipFree(m21));
	testCUDA(hipFree(m22));

	testCUDA(hipFree(SIG));

	testCUDA(hipFree(Alpha));

	testCUDA(hipFree(X));
	testCUDA(hipFree(Y));
	testCUDA(hipFree(Z));	

	testCUDA(hipFree(val1));
	testCUDA(hipFree(var1));	
	testCUDA(hipFree(val2));
	testCUDA(hipFree(var2));	

	testCUDA(hipFree(EZ));
	testCUDA(hipFree(VZ));	
	testCUDA(hipFree(EZt));
	testCUDA(hipFree(VZt));	
	testCUDA(hipFree(EY));
	testCUDA(hipFree(VY));
	testCUDA(hipFree(EYt));
	testCUDA(hipFree(VYt));		
}


////////////////////////////////////////////////////////////
// Memory allocation for regressed values
////////////////////////////////////////////////////////////
void RegMalloc(AP ap)
{
	int NI = ap.NI;
	testCUDA(hipMalloc(&Matcorr, NbOuter*Dim*Dim * sizeof(float)));
	
	testCUDA(hipHostAlloc(&GammaZ, NbOuter*Dim*Dim*sizeof(float), hipHostMallocMapped));
	testCUDA(hipHostAlloc(&GammaY, ((NI*(NI-1))/2)*NbOuter*Dim *sizeof(float), hipHostMallocMapped));
	testCUDA(hipHostAlloc(&Cst, ((NI*(NI - 1))/2)*NbOuter * sizeof(float), hipHostMallocMapped));
}

////////////////////////////////////////////////////////////
// Memory deallocation for regressed values
////////////////////////////////////////////////////////////
void RegFree(void)
{
	testCUDA(hipFree(Matcorr));
	testCUDA(hipHostFree(GammaZ));
	testCUDA(hipHostFree(GammaY));	
	testCUDA(hipHostFree(Cst));
}

////////////////////////////////////////////////////////////
// Memory allocation for inner trajectories
////////////////////////////////////////////////////////////
void InMalloc(AP ap)
{
	testCUDA(hipMalloc(&XI, 2*Dim*NbOuter*NbInner * sizeof(float)));
}

////////////////////////////////////////////////////////////
// Memory deallocation for inner trajectories
////////////////////////////////////////////////////////////
void InFree(void)
{
	testCUDA(hipFree(XI));
}